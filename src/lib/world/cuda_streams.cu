#include <hip/hip_runtime.h>
//#include <world/cuda_streams.h>

#include <hipblas.h>
//#define NUM_STREAMS 16
//cudaStream_t streams[NUM_STREAMS];

extern "C" void * cublashandle_create(){
    hipblasHandle_t * handle = new hipblasHandle_t;
    hipblasCreate(handle);
    void * h = (void *)handle;
    return h;
}

extern "C" void cublashandle_destroy(void * h){
    hipblasHandle_t * handle = (hipblasHandle_t*)h;
    hipblasDestroy(*handle);
    delete handle;
}

extern "C" void ** streams_initialize(unsigned int streams){
    unsigned int i;
    void ** cast_streams = new void*[streams];
    for (i = 0; i < streams; i++){
        hipStream_t * stream = new hipStream_t;
        hipStreamCreate(stream);
        cast_streams[i] = (void *)stream;
    }
    return cast_streams;
}


extern "C" void streams_destroy(void ** cast_streams, unsigned int streams){
    unsigned int i;
    for (i = 0; i < streams; i++){
        hipStream_t * stream = (hipStream_t *)cast_streams[i];
        hipStreamDestroy(*stream);
        delete stream;
    }
}

extern "C" void streams_synchronize(void ** cast_streams, unsigned int streams){
    unsigned int i;
    for (i = 0; i < streams; i++){
        hipStream_t * stream = (hipStream_t *)cast_streams[i];
        hipStreamSynchronize(*stream);
    }

}
